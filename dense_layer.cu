#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dense_layer.h"

// Matrix multiplication kernel with bias addition for a dense layer
__global__ void forward_kernel(float* X, float* W, float* b, float* Y, int batch, int in_size, int out_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // sample index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // output neuron index

    if (row < batch && col < out_size) {
        float sum = 0.0f;
        for (int i = 0; i < in_size; ++i) {
            sum += X[row * in_size + i] * W[i * out_size + col];
        }
        Y[row * out_size + col] = sum + b[col];
    }
}

// ReLU activation kernel
__global__ void relu_kernel(float* A, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        A[idx] = fmaxf(0.0f, A[idx]);
    }
}

__global__ void relu_backward_kernel(
    float*       dY_hidden,  // [batch × HIDDEN_DIM]
    const float* hidden,     // pre‐ReLU activations [batch × HIDDEN_DIM]
    int total
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total && hidden[idx] <= 0.0f) {
        dY_hidden[idx] = 0.0f;
    }
}

// Softmax kernel, operates per row (sample)
__global__ void softmax_kernel(float* input, float* output, int batch, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // row index
    if (i >= batch) return;

    // Find max for numerical stability
    float max_val = input[i * dim];
    for (int j = 1; j < dim; ++j) {
        max_val = fmaxf(max_val, input[i * dim + j]);
    }

    // Compute softmax
    float sum_exp = 0.0f;
    for (int j = 0; j < dim; ++j) {
        output[i * dim + j] = expf(input[i * dim + j] - max_val);
        sum_exp += output[i * dim + j];
    }
    for (int j = 0; j < dim; ++j) {
        output[i * dim + j] /= sum_exp;
    }
}

// Backpropagation for dense layer (computes dW and db)
__global__ void backward_kernel(float* dY, float* X, float* dW, float* db, int batch, int in_size, int out_size) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // input feature index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // output neuron index

    if (i < in_size && j < out_size) {
        float grad = 0.0f;
        for (int n = 0; n < batch; ++n) {
            grad += X[n * in_size + i] * dY[n * out_size + j];
        }
        dW[i * out_size + j] = grad / batch;
    }

    // Compute db for each output neuron (only once per output neuron)
    if (i == 0 && j < out_size) {
        float bias_grad = 0.0f;
        for (int n = 0; n < batch; ++n) {
            bias_grad += dY[n * out_size + j];
        }
        db[j] = bias_grad / batch;
    }
}

__global__ void hidden_grad_kernel(
    const float* __restrict__ dY,  
    const float* __restrict__ W2,  
    float*             dY_hidden,   
    int batch,
    int hiddenSize,
    int outputSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch * hiddenSize;
    if (idx >= total) return;

    int n = idx / hiddenSize;   // sample index
    int h = idx % hiddenSize;   // hidden‐unit index

    const float* dY_row = dY       + n * outputSize;
    const float* W2_row = W2       + h * outputSize;

    float sum = 0.0f;
    #pragma unroll
    for (int k = 0; k < outputSize; ++k) {
        sum += dY_row[k] * W2_row[k];
    }
    dY_hidden[idx] = sum;
}



__global__ void loss_gradient_kernel(const float* probs, const int* labels, float* dY, float* loss, int batch, int classes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch) return;

    int label = labels[i];
    float sample_loss = -logf(probs[i * classes + label] + 1e-8f); // avoid log(0)
    loss[i] = sample_loss;

    for (int j = 0; j < classes; ++j) {
        dY[i * classes + j] = probs[i * classes + j] - (label == j ? 1.0f : 0.0f);
    }
}

__global__ void accuracy_kernel(const float* probs, const int* labels, int* correct, int batch, int classes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch) return;

    int max_idx = 0;
    float max_val = probs[i * classes];
    for (int j = 1; j < classes; ++j) {
        float val = probs[i * classes + j];
        if (val > max_val) {
            max_val = val;
            max_idx = j;
        }
    }
    correct[i] = (max_idx == labels[i]) ? 1 : 0;
}



// Launches forward kernel
void dense_forward(float* X, float* W, float* b, float* Y, int batch, int in_size, int out_size) {
    dim3 block(16, 16);
    dim3 grid((out_size + 15) / 16, (batch + 15) / 16);
    forward_kernel<<<grid, block>>>(X, W, b, Y, batch, in_size, out_size);
    hipDeviceSynchronize();
}

// Launches backward kernel
void dense_backward(float* dY, float* X, float* dW, float* db, int batch, int in_size, int out_size) {
    dim3 block(16, 16);
    dim3 grid((out_size + 15) / 16, (in_size + 15) / 16);
    backward_kernel<<<grid, block>>>(dY, X, dW, db, batch, in_size, out_size);
    hipDeviceSynchronize();
}

// Launches ReLU activation kernel
void relu_forward(float* A, int total) {
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    relu_kernel<<<gridSize, blockSize>>>(A, total);
    hipDeviceSynchronize();
}

// Launches softmax kernel
void softmax_forward(float* input, float* output, int batch, int dim) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    softmax_kernel<<<gridSize, blockSize>>>(input, output, batch, dim);
    hipDeviceSynchronize();
}

void compute_loss_and_gradient_cuda(const float* probs, const int* labels, float* dY, float* loss_array, int batch, int classes) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    loss_gradient_kernel<<<gridSize, blockSize>>>(probs, labels, dY, loss_array, batch, classes);
    hipDeviceSynchronize();
}

void compute_accuracy_cuda(const float* probs, const int* labels, int* correct_array, int batch, int classes) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    accuracy_kernel<<<gridSize, blockSize>>>(probs, labels, correct_array, batch, classes);
    hipDeviceSynchronize();
}

void hidden_grad(
    const float* dY, const float* W2, float* dY_hidden,
    int batch, int hiddenSize, int outputSize
) {
    int total     = batch * hiddenSize;
    int blockSize = 256;
    int gridSize  = (total + blockSize - 1) / blockSize;
    hidden_grad_kernel<<<gridSize, blockSize>>>(
        dY, W2, dY_hidden, batch, hiddenSize, outputSize
    );
    hipDeviceSynchronize();
}

void relu_backward(float* dY_hidden, const float* hidden, int total) {
    int blockSize = 256;
    int gridSize  = (total + blockSize - 1) / blockSize;
    relu_backward_kernel<<<gridSize, blockSize>>>(dY_hidden, hidden, total);
    hipDeviceSynchronize();
}