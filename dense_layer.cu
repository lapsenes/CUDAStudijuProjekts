#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// leaky ReLU layer
__global__ void leaky_relu_kernel(float* A, int total, float alpha) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // compute global thread index
    if (idx < total) { // bound check
        A[idx] = (A[idx] > 0.0f) ? A[idx] : alpha * A[idx]; // keeps the positive value, multiplies the negative value with a small alpha
    }
}

// Forward pass kernel for dense layer
__global__ void forward_kernel(float* X, float* W, float* b, float* Y, int batch, int in_size, int out_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < batch && col < out_size) {
        float sum = 0.0f;
        for (int i = 0; i < in_size; ++i) {
            sum += X[row * in_size + i] * W[i * out_size + col];
        }
        Y[row * out_size + col] = sum + b[col];
    }
}

// Leaky ReLU backward kernel
__global__ void leaky_relu_backward_kernel(float* dY, float* hidden, int total, float alpha) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        if (hidden[idx] <= 0.0f) {
            dY[idx] *= alpha;  // If input <= 0, scale the gradient by alpha
        }
    }
}

// Backward pass kernel for dense layer (computing dW and db)
__global__ void backward_kernel(float* dY, float* X, float* dW, float* db, int batch, int in_size, int out_size, bool use_leaky_relu = false, float alpha = 0.01f) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // input feature index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // output neuron index

    if (i < in_size && j < out_size) {
        float grad = 0.0f;
        for (int n = 0; n < batch; ++n) {
            grad += X[n * in_size + i] * dY[n * out_size + j];
        }
        dW[i * out_size + j] = grad / batch;
    }

    if (i == 0 && j < out_size) {
        float bias_grad = 0.0f;
        for (int n = 0; n < batch; ++n) {
            bias_grad += dY[n * out_size + j];
        }
        db[j] = bias_grad / batch;
    }

    // If using Leaky ReLU, adjust the gradient calculation
    if (use_leaky_relu && i < in_size && j < out_size) {
        // Leaky ReLU gradient adjustment: if the input was <= 0, multiply the gradient by alpha
        if (X[i] <= 0.0f) {
            dY[i] *= alpha;  // Adjust the gradient for the dead neurons (Leaky ReLU)
        }
    }
}

// Softmax kernel (used after the output layer)
__global__ void softmax_kernel(float* input, float* output, int batch, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i >= batch) return;

    float max_val = input[i * dim];
    for (int j = 1; j < dim; ++j) {
        max_val = fmaxf(max_val, input[i * dim + j]);
    }

    float sum_exp = 0.0f;
    for (int j = 0; j < dim; ++j) {
        output[i * dim + j] = expf(input[i * dim + j] - max_val);
        sum_exp += output[i * dim + j];
    }
    for (int j = 0; j < dim; ++j) {
        output[i * dim + j] /= sum_exp;
    }
}

// Loss gradient kernel for cross-entropy loss and backpropagation
__global__ void loss_gradient_kernel(const float* probs, const int* labels, float* dY, float* loss, int batch, int classes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch) return;

    int label = labels[i];
    float sample_loss = -logf(probs[i * classes + label] + 1e-8f); // Log with small epsilon for numerical stability
    loss[i] = sample_loss;

    for (int j = 0; j < classes; ++j) {
        dY[i * classes + j] = probs[i * classes + j] - (label == j ? 1.0f : 0.0f);
    }
}

// Accuracy kernel
__global__ void accuracy_kernel(const float* probs, const int* labels, int* correct, int batch, int classes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= batch) return;

    int max_idx = 0;
    float max_val = probs[i * classes];
    for (int j = 1; j < classes; ++j) {
        float val = probs[i * classes + j];
        if (val > max_val) {
            max_val = val;
            max_idx = j;
        }
    }
    correct[i] = (max_idx == labels[i]) ? 1 : 0;
}


// Kernel: dY_hidden[n, h] = sum_k dY[n, k] * W2[h, k]
__global__ void hidden_grad_kernel(
    const float* __restrict__ dY,        // [batchSize, outputSize]
    const float* __restrict__ W2,        // [hiddenSize, outputSize]
    float*             dY_hidden,        // [batchSize, hiddenSize]
    int batchSize,
    int hiddenSize,
    int outputSize
) {
    int idx   = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batchSize * hiddenSize;
    if (idx >= total) return;

    int n = idx / hiddenSize;  // sample index
    int h = idx % hiddenSize;  // hidden feature index

    // pointers to row n of dY, row h of W2
    const float* dY_row = dY       + n * outputSize;
    const float* W2_row = W2       + h * outputSize;

    float sum = 0.0f;
    #pragma unroll
    for (int k = 0; k < outputSize; ++k) {
        sum += dY_row[k] * W2_row[k];
    }

    dY_hidden[idx] = sum;
}







// Forward pass function for dense layer
void dense_forward(float* X, float* W, float* b, float* Y, int batch, int in_size, int out_size) {
    dim3 block(16, 16);
    dim3 grid((out_size + 15) / 16, (batch + 15) / 16);
    forward_kernel<<<grid, block>>>(X, W, b, Y, batch, in_size, out_size);
    hipDeviceSynchronize();
}

// Backward pass function for dense layer
void dense_backward(float* dY, float* X, float* dW, float* db, int batch, int in_size, int out_size, bool use_leaky_relu = false, float alpha = 0.01f) {
    dim3 block(16, 16);
    dim3 grid((out_size + 15) / 16, (in_size + 15) / 16);
    backward_kernel<<<grid, block>>>(dY, X, dW, db, batch, in_size, out_size, use_leaky_relu, alpha);
    hipDeviceSynchronize();
}

// Leaky ReLU forward function
void leaky_relu_forward(float* A, int total, float alpha = 0.01f) {
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    leaky_relu_kernel<<<gridSize, blockSize>>>(A, total, alpha);
    hipDeviceSynchronize();
}

// Softmax forward function
void softmax_forward(float* input, float* output, int batch, int dim) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    softmax_kernel<<<gridSize, blockSize>>>(input, output, batch, dim);
    hipDeviceSynchronize();
}

// Loss and gradient computation function
void compute_loss_and_gradient_cuda(const float* probs, const int* labels, float* dY, float* loss_array, int batch, int classes) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    loss_gradient_kernel<<<gridSize, blockSize>>>(probs, labels, dY, loss_array, batch, classes);
    hipDeviceSynchronize();
}

// Accuracy computation function
void compute_accuracy_cuda(const float* probs, const int* labels, int* correct_array, int batch, int classes) {
    int blockSize = 256;
    int gridSize = (batch + blockSize - 1) / blockSize;
    accuracy_kernel<<<gridSize, blockSize>>>(probs, labels, correct_array, batch, classes);
    hipDeviceSynchronize();
}

// Leaky ReLU backward function
void leaky_relu_backward(float* dY, float* hidden, int total, float alpha = 0.01f) {
    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;
    leaky_relu_backward_kernel<<<gridSize, blockSize>>>(dY, hidden, total, alpha);
    hipDeviceSynchronize();
}


void hidden_grad(const float* dY, const float* W2, float* dY_hidden, int batchSize, int hiddenSize, int outputSize) {
    int total     = batchSize * hiddenSize;
    int blockSize = 256;
    int gridSize  = (total + blockSize - 1) / blockSize;
    hidden_grad_kernel<<<gridSize, blockSize>>>(dY, W2, dY_hidden, batchSize, hiddenSize, outputSize);
    hipDeviceSynchronize();
}